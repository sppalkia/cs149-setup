#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cstdio>

#define CUDA_SAFE_CALL(expr)				\
	{						\
		hipError_t err = (expr);		\
		if (err != hipSuccess)			\
		{					\
			printf("Cuda error: %s\n", hipGetErrorString(err));	\
			exit(1);			\
		}					\
	}

int main(void)
{
  int deviceCount;
  CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
  printf("There are %d devices.\n\n", deviceCount);
  int device;
  for(device = 0; device<deviceCount; device++)
  {
    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, device));
    if(device == 0)
    {
      if(deviceProp.major == 9999 && deviceProp.minor == 9999)
        printf("There is no device supporting CUDA.\n");
      else if(deviceCount == 1)
        printf("This is 1 device supporting CUDA.\n");
      else
        printf("There are %d devices supporting CUDA.\n", deviceCount);
    }
    printf("Device %d is called %s\n", device, deviceProp.name);

    printf("\tDevice Properties:\n");
    printf("\t\tHas timeout enabled: %d\n",deviceProp.kernelExecTimeoutEnabled);
    printf("\t\tECC enabled: %d\n",deviceProp.ECCEnabled);
    printf("\t\tClock rate %ld Hz\n",long(deviceProp.clockRate)*1000);
    printf("\t\tCompute capability: %d.%d\n",deviceProp.major,deviceProp.minor);
    printf("\t\tCompute mode: %d\n",deviceProp.computeMode);
    printf("\t\tConcurrent kernels: %d\n", deviceProp.concurrentKernels);
    printf("\t\tIntegrated device: %d\n",deviceProp.integrated);
    printf("\t\tSupports unified addressing: %d\n",deviceProp.unifiedAddressing);

    printf("\tCompute Properties:\n");
    printf("\t\tNumber of SMs: %d\n",deviceProp.multiProcessorCount);
    printf("\t\tWarp size: %d threads\n",deviceProp.warpSize);
    printf("\t\tMaximum threads per CTA: %d\n",deviceProp.maxThreadsPerBlock);
    printf("\t\tMaximum threads per SM: %d\n",deviceProp.maxThreadsPerMultiProcessor);
    printf("\t\tMaximum warps per CTA: %d\n",(deviceProp.maxThreadsPerBlock/deviceProp.warpSize));
    printf("\t\tMaximum warps per SM: %d\n",(deviceProp.maxThreadsPerMultiProcessor/deviceProp.warpSize));
    printf("\t\tMaximum grid size: ");
    for (int i=0; i<3; i++)
      printf("%d ",deviceProp.maxGridSize[i]);
    printf("\n");
    printf("\t\tMaximum CTA size: ");
    for (int i=0; i<3; i++)
      printf("%d ",deviceProp.maxThreadsDim[i]);
    printf("\n");

    printf("\tMemory Properties:\n");
    printf("\t\tTotal global memory: %ld bytes\n",deviceProp.totalGlobalMem);
    printf("\t\tTotal constant memory: %ld bytes\n",deviceProp.totalConstMem);
    printf("\t\tL2 cache size: %d bytes\n",deviceProp.l2CacheSize);
    printf("\t\tShared memory per block: %ld bytes\n",deviceProp.sharedMemPerBlock);
    printf("\t\tRegisters per block: %d\n",deviceProp.regsPerBlock);
    size_t stackSize,pfSize,heapSize;
    CUDA_SAFE_CALL(hipDeviceGetLimit(&stackSize, hipLimitStackSize));
    CUDA_SAFE_CALL(hipDeviceGetLimit(&pfSize, hipLimitPrintfFifoSize));
    CUDA_SAFE_CALL(hipDeviceGetLimit(&heapSize, hipLimitMallocHeapSize));
    printf("\t\tStack size per thread: %ld bytes\n",stackSize);
    printf("\t\tMalloc heap size: %ld bytes\n",heapSize);
    printf("\t\tPrintf buffer size: %ld bytes\n",pfSize);
    printf("\t\tMemory bus width: %d bits\n",deviceProp.memoryBusWidth); 
    printf("\t\tMemory pitch: %ld bytes\n",deviceProp.memPitch);

    printf("\tPCI-E Xfer Properties:\n");
    printf("\t\tNumber of asynchronous engines (async-copy enabled): %d\n",deviceProp.asyncEngineCount);
    printf("\t\tCan Map Host Memory: %d\n",deviceProp.canMapHostMemory);
    printf("\t\tPCI device ID: %d\n",deviceProp.pciDeviceID);
    printf("\t\tPCI bus ID: %d\n",deviceProp.pciBusID);
    
    printf("\n\n");
  }

  return 0;
}
